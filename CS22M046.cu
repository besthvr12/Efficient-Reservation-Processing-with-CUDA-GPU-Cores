#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************
/*
Approach : Here in this question we need to find number of successful request at a center and number of unsuccesful request at center
and Number of success request and Number of failure from total request. Here first I have create a slot array which will store all the slots
and their corresponding capacity , we have also calculate the offset for every capacity and facilities. After creating the slots array we have call a kernel.
Now in kernel call for every thread in parallel we have done the parallel computation center and facility wise. 
At last we have store the computation.
*/
__global__ void all(int N,int R, int* d_center, int* d_facility, int* d_facids, int* d_capacity, int* d_offset, int *d_reqid, int *d_reqcen, int *d_reqfac, int *d_reqstart, int *d_reqslots, int* d_totalreqs, int* d_succesreqs, int* d_slots, int* d_success, int* d_failure,int np)
{
    int blockId = blockIdx.x * blockDim.x;
    int id = blockId + threadIdx.x; //Calculating the threadId
    bool flag;
    if(id < np) // If our id is smaller then N*max_P
    {
      
        for(int k = 0; k < R; k++)// Then traverse for maximum all the resources and do computation only when its mataches its center and facility ids
        {
            flag = true;
            int cenfac = d_reqcen[k] * 30;
            int dfac = d_reqfac[k];
            int res = cenfac + dfac ;// If we find then that our reqcen and reqfac matches then we can do the computation
            int count = 0; 
            if(res== id)
            {
                int temp = d_reqstart[k] + d_reqslots[k];
                if(temp<=25) // It will work at maximum for 25 if reqstart and reqslots is greater then 25 it means it can be given access so just make it false
                {   
                    cenfac  = d_reqcen[k] * 30;
                    dfac = d_reqfac[k];
                    int cenfacres = cenfac+ dfac;
                    int slotstart = (cenfacres) * 24 + d_reqstart[k];  // Starting position for every slot
                    for(int i = 0; i < d_reqslots[k]; i++) // It will traverse at maximum of number of requested slots
                    {
                        if(d_slots[ slotstart + i-1]>0)//If this slots is not empty then we can decrease the capacity
                        {              
                              atomicSub(&d_slots[slotstart + i-1], 1); // Now decrease the capacity
                        }
                        else
                        {
                            int j=0;
                            while(j<i)
                            {
                                atomicAdd(&d_slots[slotstart + j-1], 1); // Now for the request which slots fails, we need to increase the slots in which it has earlier decreses
                                j++;
                            }
                            flag = false;
                            break;
                        }
                        
                    }
                }
                else
                {
                    count++;
                    flag = false;
                }

    
                if (flag) {
                  atomicAdd(&d_success[0], 1); //Atomic add the number of success request
                  atomicAdd(&d_succesreqs[d_reqcen[k]], 1); // Atomic add the number of success request at a center
        }     else {
                  atomicAdd(&d_failure[0], 1); // Atomic add the number of failures at a center
        }
            }

            
        }

        
    }

}


//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs,*slots;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	    int *offset = (int *) malloc ( (N) * sizeof (int) ); /// This will store the offset from where next capacity is starting and so on
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N *  sizeof (int));  // stores capacities of each facility for every computer centre 
    slots=(int*)malloc(max_P * N * 24 * sizeof (int));  // It will store the slots on which we can work for every facility there are 24 slots available and there can be maximum of 
    // 30 facility at a center

    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0,k3=0;
    for(int i=0;i<N;i++)
    {
      k3 = 0;
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      offset[i]=k1;// Storing offset for every center how many facilities its need to move
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        for(int k4=0;k4<24;k4++){
            int index = centre[i]*30*24;
            slots[index + k3]=capacity[k2];
            k3++;
        }
        k2++; 
        
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	  int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }

     int np = max_P * N;
   	int *d_center, *d_facility, *d_facids , *d_capacity , *d_offset , *d_reqid, *d_reqcen, *d_reqfac, *d_reqstart, *d_reqslots, *d_slots, *d_totalreqs, *d_succesreqs,*d_success,*d_failure;
    hipMalloc(&d_center, (N) * sizeof(int)); // It will store the centers in device
    hipMemcpy(d_center, centre, N * sizeof(int), hipMemcpyHostToDevice);

	  hipMalloc(&d_facility, (N) * sizeof(int));//It will store the facility in the device
    hipMemcpy(d_facility, facility, N * sizeof(int), hipMemcpyHostToDevice);

  	hipMalloc(&d_facids, (max_P*N) * sizeof(int)); // It will store the facids in the device
    hipMemcpy(d_facids, fac_ids, max_P * N * sizeof(int), hipMemcpyHostToDevice);

	  hipMalloc(&d_capacity, (max_P*N) * sizeof(int)); // It will store the capacity for every facility of the centers
	  hipMemcpy(d_capacity, capacity, max_P * N * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_offset, (N) * sizeof(int)); // It will store the offset
	  hipMemcpy(d_offset, offset,  N * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_reqid, (R) * sizeof(int)); // It will store the Request Id
    hipMemcpy(d_reqid, req_id, R * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_reqcen, (R) * sizeof(int)); // It will store the Request center
	  hipMemcpy(d_reqcen, req_cen, R * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_reqfac, (R) * sizeof(int)); // It will store the request facility Id
	  hipMemcpy(d_reqfac, req_fac, R * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_reqstart, (R) * sizeof(int)); // It will store the starting time of request
    hipMemcpy(d_reqstart, req_start, R * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_reqslots, (R) * sizeof(int)); // It will store the number of required slots
    hipMemcpy(d_reqslots, req_slots, R * sizeof(int), hipMemcpyHostToDevice);

	  hipMalloc(&d_slots, (max_P*N*24) * sizeof(int)); // It will store the for every facility and center slots
	  hipMemcpy(d_slots, slots, N*max_P*24 * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_totalreqs, (N) * sizeof(int)); // It will store the total request at the center
    hipMemcpy(d_totalreqs, tot_reqs, N * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_succesreqs, (N) * sizeof(int)); // It will store the number of success request from center
    hipMalloc(&d_succesreqs, (N) * sizeof(int)); 

    hipMalloc(&d_success,sizeof(int)); // It will return number of successful request
    hipMemcpy(d_success, &success, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_failure,  sizeof(int)); // It will return number of failure
    hipMemcpy(d_failure, &fail,sizeof(int), hipMemcpyHostToDevice);  
 
    
    // Output
    //*********************************
    // Call the kernels here
    //********************************
  
    all<<<np, 1024>>>(N, R, d_center, d_facility, d_facids, d_capacity, d_offset, d_reqid, d_reqcen, d_reqfac, d_reqstart, d_reqslots, d_totalreqs, d_succesreqs, d_slots, d_success, d_failure,np); 
    
    hipMemcpy(succ_reqs,d_succesreqs, N * sizeof(int), hipMemcpyDeviceToHost);  
    hipMemcpy(&fail, d_failure, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&success, d_success, sizeof(int), hipMemcpyDeviceToHost);
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");
    
    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
      fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}